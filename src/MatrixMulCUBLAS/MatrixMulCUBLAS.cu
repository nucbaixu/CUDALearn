#include "hip/hip_runtime.h"

// Matrix multiplication: C = A * B.
// Host code.
//
// This sample implements matrix multiplication as described in Chapter 3
// of the programming guide and uses the CUBLAS library to demonstrate
// the best performance.

// SOME PRECAUTIONS:
// IF WE WANT TO CALCULATE ROW-MAJOR MATRIX MULTIPLY C = A * B,
// WE JUST NEED CALL CUBLAS API IN A REVERSE ORDER: cublasSegemm(B, A)!
// The reason is explained as follows:

// CUBLAS library uses column-major storage, but C/C++ use row-major storage.
// When passing the matrix pointer to CUBLAS, the memory layout alters from
// row-major to column-major, which is equivalent to an implicit transpose.
// Utilities and system includes


#include <helper_string.h>  // helper for shared functions common to CUDA Samples

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif

typedef struct _matrixSize      // Optional Command-line multiplier for matrix sizes
{
    unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
} sMatrixSize;

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on CPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
void matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0 ; i < hA; i++)
    {
        for(unsigned int j = 0; j < wB; j++)
        {
            float  f_sum = 0;

            for(int k = 0; k < wA;k++)
            {
                f_sum += A[i * wA + k] * B[k * wB + j];
            }

            C[i * wB + j] = f_sum;
        }
    }
}

// Allocates a matrix with random float entries.
void randomInit(float *data,  unsigned int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = float( rand())  / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;

    for (j = 0; j < height; j++)
    {
        if (error_count < iListLength)
        {
            printf("\n  Row %d:\n", j);
        }

        for (i = 0; i < width; i++)
        {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);

            if (fDiff > fListTol)
            {
                if (error_count < iListLength)
                {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }

                error_count++;
            }
        }
    }

    printf(" \n  Total Errors = %d\n", error_count);
}

void initializeCUDA(int argc, char **argv, int &devID, int &iSizeMultiple, sMatrixSize &matrix_size)
{
    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    hipError_t error;
    devID = 0;

    devID = findCudaDevice(argc, (const char **)argv);

    if (checkCmdLineFlag(argc, (const char **)argv, "sizemult"))
    {
        iSizeMultiple = getCmdLineArgumentInt(argc, (const char **)argv, "sizemult");
    }

    iSizeMultiple = min(iSizeMultiple, 10);
    iSizeMultiple = max(iSizeMultiple, 1);

    hipDeviceProp_t deviceProp{};

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

    int block_size = 32;

    matrix_size.uiWA = 3 * block_size * iSizeMultiple;
    matrix_size.uiHA = 4 * block_size * iSizeMultiple;
    matrix_size.uiWB = 2 * block_size * iSizeMultiple;
    matrix_size.uiHB = 3 * block_size * iSizeMultiple;
    matrix_size.uiWC = 2 * block_size * iSizeMultiple;
    matrix_size.uiHC = 4 * block_size * iSizeMultiple;

    printf("MatrixA(%u,%u), MatrixB(%u,%u), MatrixC(%u,%u)\n",
           matrix_size.uiHA, matrix_size.uiWA,
           matrix_size.uiHB, matrix_size.uiWB,
           matrix_size.uiHC, matrix_size.uiWC);

    if( matrix_size.uiWA != matrix_size.uiHB ||
    matrix_size.uiHA != matrix_size.uiHC ||
    matrix_size.uiWB != matrix_size.uiWC)
    {
        printf("ERROR: Matrix sizes do not match!\n");
        exit(-1);
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test matrix multiply using CUBLAS
////////////////////////////////////////////////////////////////////////////////
int matrixMultiply_V1(int argc, char **argv, int devID, sMatrixSize &matrix_size)
{
    hipDeviceProp_t deviceProp{};

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    int block_size = 32;

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = matrix_size.uiWA * matrix_size.uiHA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    auto *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = matrix_size.uiWB * matrix_size.uiHB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    auto *h_B = (float *)malloc(mem_size_B);

    // set seed for rand()
    srand(2006);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    unsigned int size_C = matrix_size.uiWC * matrix_size.uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    auto *h_C      = (float *) malloc(mem_size_C);
    auto *h_CUBLAS = (float *) malloc(mem_size_C);

    checkCudaErrors(hipMalloc((void **) &d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void **) &d_B, mem_size_B));
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &d_C, mem_size_C));

    // setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(matrix_size.uiWC / threads.x, matrix_size.uiHC / threads.y);

    // create and start timer
    printf("Computing result using CUBLAS...");

    int nIter = 30;

    //CUBLAS Version 2.0
    {
        const float alpha = 1.0f;
        const float beta = 0.0;

        hipblasHandle_t handle;
        hipEvent_t start,stop;

        checkCudaErrors(hipblasCreate(&handle));

        //preform warmup operation with cublas
        //hipblasSgemm是CUDA的cublas库的矩阵相乘函数
        //https://blog.csdn.net/u011197534/article/details/78378536
        //https://blog.csdn.net/HaoBBNuanMM/article/details/103054357
        //当我们选择HIPBLAS_OP_N时表示不转置，按列优先存储；当我们选择HIPBLAS_OP_T时表示需要转置，按行优先存储

        checkCudaErrors(hipblasSgemm(handle
                                    , HIPBLAS_OP_N
                                    , HIPBLAS_OP_N
                                    , matrix_size.uiWC
                                    , matrix_size.uiHC
                                    , matrix_size.uiWA
                                    , &alpha
                                    , d_B
                                    , matrix_size.uiWB
                                    , d_A
                                    , matrix_size.uiWA
                                    , &beta
                                    , d_C
                                    , matrix_size.uiWC));

        // Allocate CUDA events that we'll use for timing
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));

        // Record the start event
        checkCudaErrors(hipEventRecord(start, nullptr));

        for (int j = 0; j < nIter; j++)
        {
            //note cublas is column primary!
            //need to transpose the order
            checkCudaErrors(hipblasSgemm(handle
                                        , HIPBLAS_OP_N
                                        , HIPBLAS_OP_N
                                        , matrix_size.uiWC
                                        , matrix_size.uiHC
                                        , matrix_size.uiWA
                                        , &alpha
                                        , d_B
                                        , matrix_size.uiWB
                                        , d_A
                                        , matrix_size.uiWA
                                        , &beta
                                        , d_C
                                        , matrix_size.uiWC));
        }

        printf("done.\n");

        // Record the stop event
        checkCudaErrors(hipEventRecord(stop, nullptr));

        // Wait for the stop event to complete
        checkCudaErrors(hipEventSynchronize(stop));

        float msecTotal = 0.0f;
        checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

        // Compute and print the performance
        float msecPerMatrixMul = msecTotal / nIter;
        double flopsPerMatrixMul = 2.0 * (double)matrix_size.uiHC * (double)matrix_size.uiWC * (double)matrix_size.uiHB;
        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
        printf(
                "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
                gigaFlops,
                msecPerMatrixMul,
                flopsPerMatrixMul);

        // copy result from device to host
        checkCudaErrors(hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost));

        // Destroy the handle
        checkCudaErrors(hipblasDestroy(handle));

    }

    // compute reference solution
    printf("Computing result using host CPU...");
    float *reference = (float *)malloc(mem_size_C);
    matrixMulCPU(reference, h_A, h_B, matrix_size.uiHA, matrix_size.uiWA, matrix_size.uiWB);
    printf("done.\n");

    // check result (CUBLAS)
    bool resCUBLAS = sdkCompareL2fe(reference, h_CUBLAS, size_C, 1.0e-6f);

    if (resCUBLAS != true)
    {
        printDiff(reference, h_CUBLAS, matrix_size.uiWC, matrix_size.uiHC, 100, 1.0e-5f);
    }

    printf("Comparing CUBLAS Matrix Multiply with CPU results: %s\n", (true == resCUBLAS) ? "PASS" : "FAIL");

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    if (resCUBLAS == true)
    {
        return EXIT_SUCCESS;    // return value = 1
    }
    else
    {
        return EXIT_FAILURE;     // return value = 0
    }

}

template<class T>
void transpose(T* a, T* b,int rows,int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            b[j*rows+i] = a[i*cols+j];
            //b[j][i] = a[i][j];
        }
    }
}

int matrixMultiply_V2(int argc, char **argv, int devID, sMatrixSize &matrix_size)
{
    hipDeviceProp_t deviceProp{};

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    int block_size = 32;

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = matrix_size.uiWA * matrix_size.uiHA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    auto *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = matrix_size.uiWB * matrix_size.uiHB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    auto *h_B = (float *)malloc(mem_size_B);

    // set seed for rand()
    srand(2006);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    unsigned int size_C = matrix_size.uiWC * matrix_size.uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    auto *h_C      = (float *) malloc(mem_size_C);
    auto *h_CUBLAS = (float *) malloc(mem_size_C);
    auto *h_CUBLAS_T = (float *) malloc(mem_size_C);

    checkCudaErrors(hipMalloc((void **) &d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void **) &d_B, mem_size_B));
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &d_C, mem_size_C));

    // setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(matrix_size.uiWC / threads.x, matrix_size.uiHC / threads.y);

    // create and start timer
    printf("Computing result using CUBLAS...");

    int nIter = 30;

    //CUBLAS Version 2.0
    {
        const float alpha = 1.0f;
        const float beta = 0.0;

        hipblasHandle_t handle;
        hipEvent_t start,stop;

        checkCudaErrors(hipblasCreate(&handle));

        //preform warmup operation with cublas
        //hipblasSgemm是CUDA的cublas库的矩阵相乘函数
        //https://blog.csdn.net/u011197534/article/details/78378536
        //https://blog.csdn.net/HaoBBNuanMM/article/details/103054357
        //当我们选择HIPBLAS_OP_N时表示不转置，按列优先存储；当我们选择HIPBLAS_OP_T时表示需要转置，按行优先存储

        checkCudaErrors(hipblasSgemm(handle
                                    , HIPBLAS_OP_T
                                    , HIPBLAS_OP_T
                                    , matrix_size.uiHC
                                    , matrix_size.uiWC
                                    , matrix_size.uiWA
                                    , &alpha
                                    , d_A
                                    , matrix_size.uiWA
                                    , d_B
                                    , matrix_size.uiWB
                                    , &beta
                                    , d_C
                                    , matrix_size.uiHC));

        // Allocate CUDA events that we'll use for timing
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));

        // Record the start event
        checkCudaErrors(hipEventRecord(start, nullptr));

        for (int j = 0; j < nIter; j++)
        {
            //note cublas is column primary!
            //need to transpose the order
            checkCudaErrors(hipblasSgemm(handle
                                        , HIPBLAS_OP_T
                                        , HIPBLAS_OP_T
                                        , matrix_size.uiHC
                                        , matrix_size.uiWC
                                        , matrix_size.uiWA
                                        , &alpha
                                        , d_A
                                        , matrix_size.uiWA
                                        , d_B
                                        , matrix_size.uiWB
                                        , &beta
                                        , d_C
                                        , matrix_size.uiHC));

        }

        printf("done.\n");

        // Record the stop event
        checkCudaErrors(hipEventRecord(stop, nullptr));

        // Wait for the stop event to complete
        checkCudaErrors(hipEventSynchronize(stop));

        float msecTotal = 0.0f;
        checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

        // Compute and print the performance
        float msecPerMatrixMul = msecTotal / nIter;
        double flopsPerMatrixMul = 2.0 * (double)matrix_size.uiHC * (double)matrix_size.uiWC * (double)matrix_size.uiHB;
        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
        printf(
                "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
                gigaFlops,
                msecPerMatrixMul,
                flopsPerMatrixMul);

        // copy result from device to host
        checkCudaErrors(hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost));

        // Destroy the handle
        checkCudaErrors(hipblasDestroy(handle));

    }

    // compute reference solution
    printf("Computing result using host CPU...");
    float *reference = (float *)malloc(mem_size_C);
    matrixMulCPU(reference, h_A, h_B, matrix_size.uiHA, matrix_size.uiWA, matrix_size.uiWB);
    printf("done.\n");

    //transpose resCUBLAS
    transpose<float>(h_CUBLAS,h_CUBLAS_T,matrix_size.uiWC,matrix_size.uiHC);

    // check result (CUBLAS)
    bool resCUBLAS = sdkCompareL2fe(reference, h_CUBLAS_T, size_C, 1.0e-6f);
    //transpose<float,float>(h_CUBLAS,h_CUBLAS_T,matrix_size.uiHC,matrix_size.uiWC);

    if (resCUBLAS != true)
    {
        printDiff(reference, h_CUBLAS_T, matrix_size.uiWC, matrix_size.uiHC, 100, 1.0e-5f);
    }

    printf("Comparing CUBLAS Matrix Multiply with CPU results: %s\n", (true == resCUBLAS) ? "PASS" : "FAIL");

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_CUBLAS_T);
    free(reference);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    if (resCUBLAS == true)
    {
        return EXIT_SUCCESS;    // return value = 1
    }
    else
    {
        return EXIT_FAILURE;     // return value = 0
    }

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("[Matrix Multiply CUBLAS] - Starting...\n");

    int devID = 0, sizeMult = 5;
    sMatrixSize matrix_size;

    initializeCUDA(argc, argv, devID, sizeMult, matrix_size);

    int matrix_result = matrixMultiply_V2(argc, argv, devID, matrix_size);

    return matrix_result;
}
























