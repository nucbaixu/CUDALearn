#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

const char *sSampleName = "simpleStreams";

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(const int* a, const int *b, int*c)
{
    unsigned int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadID < N)
    {
        c[threadID] = (a[threadID] + b[threadID]) / 2;
    }
}

int TestNoStream()
{
    //启动计时器
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, nullptr);

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    //在GPU上分配内存
    hipMalloc((void**)&dev_a, FULL_DATA_SIZE * sizeof(int));
    hipMalloc((void**)&dev_b, FULL_DATA_SIZE * sizeof(int));
    hipMalloc((void**)&dev_c, FULL_DATA_SIZE * sizeof(int));

    //在CPU上分配可分页内存
    host_a = (int*)malloc(FULL_DATA_SIZE * sizeof(int));
    host_b = (int*)malloc(FULL_DATA_SIZE * sizeof(int));
    host_c = (int*)malloc(FULL_DATA_SIZE * sizeof(int));

    //主机上的内存赋值
    for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
        host_a[i] = i;
        host_b[i] = FULL_DATA_SIZE - i;
    }

    //从主机到设备复制数据
    hipMemcpy(dev_a, host_a, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);

    kernel <<<FULL_DATA_SIZE / 1024, 1024 >>> (dev_a, dev_b, dev_c);

    //数据拷贝回主机
    hipMemcpy(host_c, dev_c, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    //计时结束
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "use time： " << elapsedTime << std::endl;

    //输出前10个结果
    for (int i = 0; i < 10; i++)
    {
        std::cout << host_c[i] << std::endl;
    }

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

int TestWithStream()
{
    //get cuda prop
    hipDeviceProp_t prop{};
    int deviceID;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop,deviceID);

    //if device support overlap function
    if(!prop.deviceOverlap)
    {
        std::cout<<"No device will handle overlaps.so no speed up from stream."<<std::endl;
        return 0;
    }

    //start timer
    hipEvent_t start,stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, nullptr);

    //creat stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    //在GPU上分配内存
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    //stream need host paned memory
    hipHostAlloc((void**)&host_a,FULL_DATA_SIZE*sizeof(int),hipHostMallocDefault);
    hipHostAlloc((void**)&host_b,FULL_DATA_SIZE*sizeof(int),hipHostMallocDefault);
    hipHostAlloc((void**)&host_c,FULL_DATA_SIZE*sizeof(int),hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
        host_a[i] = i;
        host_b[i] = FULL_DATA_SIZE - i;
    }

    for(int i = 0; i < FULL_DATA_SIZE; i+= N)
    {
        hipMemcpyAsync(dev_a,host_a+i,N * sizeof(int),hipMemcpyHostToDevice,stream);
        hipMemcpyAsync(dev_b,host_b+i,N * sizeof(int),hipMemcpyHostToDevice,stream);
        kernel<<<N / 1024,1024,0,stream>>>(dev_a,dev_b,dev_c);
        hipMemcpyAsync(host_c + i,dev_c,N*sizeof (int),hipMemcpyDeviceToHost,stream);
    }

    // wait until gpu execution finish
    hipStreamSynchronize(stream);

    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "use time:" << elapsedTime << std::endl;
    for (int i = 0; i < 10; i++)
    {
        std::cout << host_c[i] << std::endl;
    }

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipStreamDestroy(stream);

    return  0;
}

int main()
{
    TestNoStream();
    TestWithStream();
    return 0;
}